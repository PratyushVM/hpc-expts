#include<stdio.h>
#include<hip/hip_runtime.h>
#define N 1     //shift/key of cipher
 
__global__ void encrypt(char *a)
{
    a[threadIdx.x]+=N;
    if(a[threadIdx.x]>122)
    a[threadIdx.x]=200-a[threadIdx.x];
}

__global__ void decrypt(char *a)
{
    a[threadIdx.x]-=N;
    if(a[threadIdx.x]<97)
    a[threadIdx.x]=200-a[threadIdx.x];
}

int main()
{
    char a[]="teststring",*b,*c;
    c=(char*)malloc(sizeof(a));
    hipMalloc(&b,sizeof(a));
    printf("Message : %s\n",a);
    hipMemcpy(b,a,sizeof(a),hipMemcpyHostToDevice);
    encrypt<<<1,sizeof(a)-1>>>(b);
    hipMemcpy(c,b,sizeof(a),hipMemcpyDeviceToHost);
    printf("Encrypted message is : %s\n",c);
    decrypt<<<1,sizeof(a)-1>>>(b);
    hipMemcpy(c,b,sizeof(a),hipMemcpyDeviceToHost);
    printf("After decrypting, message is : %s\n",c);
}