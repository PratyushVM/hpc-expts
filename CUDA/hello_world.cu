#include<stdio.h>
#include<hip/hip_runtime.h>

__global__ void dkernel(){
   printf("hello world from %d\n",threadIdx.x);
}

int main()
{
    dkernel<<<1,8>>>();
    hipDeviceSynchronize();
    return 0;
}
